#include "hip/hip_runtime.h"
#include <sstream>

#include "guest.cuh"

__global__ void kernel() {}

std::string guest(const std::string& host) {
    // @todo(When I have access to a GPU) Implement in a kernel
    std::ostringstream oss;
    oss << "Hello " << host << ", I'm CUDA!";
    return oss.str();
}
